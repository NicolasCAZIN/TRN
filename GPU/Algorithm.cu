#include "hip/hip_runtime.h"
#include "stdafx.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Algorithm.cuh"
#include "Random.cuh"
#include "Driver.cuh"

#include <hip/hip_vector_types.h>
#include <hipcub/hipcub.hpp>

#define BLOCK_X 32
#define BLOCK_Y 32
#define warpSize 32

/*template< typename T >
void check(T result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n",
			file, line, static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
		DEVICE_RESET
			// Make sure we call CUDA Device Reset before exiting
			exit(EXIT_FAILURE);
	}
}*/


// This will output the proper CUDA error strings in the event that a CUDA host call returns an error


//#define checkCudaErrors(val)           throw_on_cuda_error ( (val), #val, __FILE__, __LINE__ )

__inline__ __device__
static bool isnan(float4 v)
{
	return isnan(v.x) || isnan(v.y) || isnan(v.z) || isnan(v.w);
}

#define warpSize 32
__inline__ __device__
static float warpReduceSum(float val)
{
	val += __shfl_down(val, 16);
	val += __shfl_down(val, 8);
	val += __shfl_down(val, 4);
	val += __shfl_down(val, 2);
	val += __shfl_down(val, 1);

	return val;
}
static inline __device__
float4 tanhf(const float4 &a)
{
	float4 b;

	b.x = tanhf(a.x);
	b.y = tanhf(a.y);
	b.z = tanhf(a.z);
	b.w = tanhf(a.w);

	return b;
}
static inline __device__
float4 expf(const float4 &a)
{
	float4 b;

	b.x = expf(a.x);
	b.y = expf(a.y);
	b.z = expf(a.z);
	b.w = expf(a.w);

	return b;
}
__global__
static void scale_kernel(float ** __restrict__ a,
	const int batch_size, const int rows, const int cols, const int stride, const float scale)

{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		for (int col = blockIdx.y * blockDim.y + threadIdx.y; col < cols; col += gridDim.y * blockDim.y)
		{
			for (int row = blockIdx.x * blockDim.x + threadIdx.x; row < rows; row += gridDim.x * blockDim.x)
			{
				a[batch][col * stride + row] *= scale;
			}
		}
	}
}


__global__
static void mean_square_error_naive_kernel(
	const int batch_size, const int rows, const int cols, const float scale,
	const float ** __restrict__ batched_predicted, const int batched_predicted_stride, 
	const float * __restrict__  expected,const int expected_stride,
	float *__restrict__  result, const int result_stride

)
{
	
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		const float *predicted = batched_predicted[batch];
		for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < rows; row += gridDim.y * blockDim.y)
		{
			float sum = 0.0f;
			for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < cols; col += gridDim.x * blockDim.x)
			{
				float d = predicted[row * batched_predicted_stride + col] - expected[row * expected_stride + col];

				sum += (d * d);
			}

			sum = warpReduceSum(sum);
			if ((threadIdx.x & 31) == 0)
				atomicAdd(&result[row * result_stride + batch], sum * scale);
		}
	}
}

__global__
static void mean_square_error_float4_kernel(
	const int batch_size, const int rows, const int cols, const float scale,
	const float ** __restrict__ batched_predicted, const int batched_predicted_stride,
	const float * __restrict__  expected, const int expected_stride,
	float *__restrict__  result, const int result_stride

)
{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		const float *predicted = batched_predicted[batch];
		for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < rows; row += gridDim.y * blockDim.y)
		{
			float sum = 0.0f;

			for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < cols; col += gridDim.x * blockDim.x)
			{
				 float4 p = reinterpret_cast<float4 *>(const_cast<float *>(&predicted[row * batched_predicted_stride]))[col];
				 float4 e = reinterpret_cast<float4 *>(const_cast<float *>(&expected[row * expected_stride]))[col];
				 float4 d = p - e;
				 float4 c = (d * d);
				sum += c.x + c.y + c.z + c.w;
			}

			sum = warpReduceSum(sum);
			if ((threadIdx.x & 31) == 0)
				atomicAdd(&result[row * result_stride + batch], sum * scale);
		}
	}
}

void compute_mean_square_error
(
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const std::size_t &batch_size,
	const float **batched_predicted, const std::size_t &batched_predicted_rows, const std::size_t &batched_predicted_cols, const std::size_t &batched_predicted_stride,
	const float *expected, const std::size_t &expected_rows, const std::size_t &expected_cols, const std::size_t &expected_stride,
	float *result, const std::size_t &result_rows, const std::size_t &result_cols, const std::size_t &result_stride
)
{
	assert(batched_predicted_rows == expected_rows);
	assert(expected_rows == result_rows);
	assert(batched_predicted_cols == expected_cols);
	assert(result_cols == batch_size);
	const float scale = 1.0f / expected_cols;
	checkCudaErrors(hipMemset2DAsync(result, result_stride * sizeof(float), 0, result_cols * sizeof(float), result_rows, stream));
	{
		dim3 grid, block;

		block.x = BLOCK_X;
		block.y = BLOCK_Y;
		block.z = 1;

		if (expected_cols >= 4)
		{
			grid.x = (expected_cols / 4 + block.x - 1) / block.x;
			grid.y = (expected_rows + block.y - 1) / block.y;
			grid.z = (batch_size + block.z - 1) / block.z;

			mean_square_error_float4_kernel << < grid, block, 0, stream >> > (
				batch_size, expected_rows, expected_cols/4,scale,
				batched_predicted, batched_predicted_stride,
				expected, expected_stride,
				result, result_stride);
		}
		else
		{
			grid.x = (expected_cols + block.x - 1) / block.x;
			grid.y = (expected_rows + block.y - 1) / block.y;
			grid.z = (batch_size + block.z - 1) / block.z;

			mean_square_error_naive_kernel << < grid, block, 0, stream >> > (
				batch_size, expected_rows, expected_cols,scale,
				batched_predicted, batched_predicted_stride,
				expected, expected_stride,
				result, result_stride);
		}

	


		checkCudaErrors(hipGetLastError());
	}
}




__global__
static void batched_sgemv_kernel
(
	const int batch_size,
	float ** __restrict__ w, const int w_rows, const int w_cols, const int w_stride,
	float ** __restrict__ x, const int x_rows, const int x_cols, const int x_stride,
	float ** __restrict__ y, const int y_rows, const int  y_cols, const int y_stride
)

{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		float *w_ = w[batch];
		float *x_ = x[batch];
		float *y_ = y[batch];

		for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < w_rows; row += gridDim.y * blockDim.y)
		{
			float sum = 0.0f;
			for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < w_cols; col += gridDim.x * blockDim.x)
			{				
				sum += dot(reinterpret_cast<float4 *>(&w_[row * w_stride])[col], reinterpret_cast<float4 *>(x_)[col]);
			}

			sum = warpReduceSum(sum);
			if ((threadIdx.x & 31) == 0)
			{
				atomicAdd(&y_[row], sum);
			}
		}
	}
}

__host__ 
static inline void batched_sgemv(
	const hipStream_t &stream,
	const std::size_t & batch_size,
	float ** w, const std::size_t &w_rows, const std::size_t &w_cols, const std::size_t &w_stride,
	float ** x, const std::size_t &x_rows, const std::size_t &x_cols, const std::size_t &x_stride,
	float ** y, const std::size_t &y_rows, const std::size_t &y_cols, const std::size_t &y_stride)
{
	assert(x_rows == 1);
	assert(y_rows == 1);
	assert(x_cols == w_cols);
	assert(y_cols == w_rows);
	dim3 grid, block;

	block.x = BLOCK_X;
	block.y = BLOCK_Y;
	block.z = 1;

	grid.x = (w_cols / 4 + block.x - 1) / block.x;
	grid.y = (w_rows + block.y - 1) / block.y;
	grid.z = (batch_size + block.z - 1) / block.z;


	batched_sgemv_kernel << <grid, block, 0, stream >> >
		(
			batch_size, w, w_rows, w_cols/4, w_stride,
			x, x_rows, x_cols/4, x_stride,
			y, y_rows, y_cols, y_stride
			);
	checkCudaErrors(hipGetLastError());

}

__global__
static void batched_reset_kernel(
	const int batch_size, const int rows, const int cols,
	 float ** __restrict__ x, const int x_stride
)
{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		float *X = x[batch];
		for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < rows; row += gridDim.y * blockDim.y)
		{
#pragma unroll 8
			for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < cols; col += gridDim.x * blockDim.x)
			{
				reinterpret_cast<float4 *>(&X[row * x_stride])[col] = make_float4(0.0f);
			}
		}
	}
}

static inline void batched_reset(const hipStream_t &stream,
	const std::size_t &batch_size, const std::size_t &rows, const std::size_t &cols,
	float **x, const std::size_t &x_stride)
{
	dim3 grid, block;
	block.x = BLOCK_X;
	block.y = BLOCK_Y;
	block.z = 1;
	grid.x = (cols / 4 + block.x - 1) / block.x;
	grid.y = (rows + block.y - 1) / block.y;
	grid.z = (batch_size + block.z - 1) / block.z;

	batched_reset_kernel << < grid, block, 0, stream >> > (
		batch_size, rows, cols/4, x, x_stride);

	checkCudaErrors(hipGetLastError());
}


__global__
static void batched_update_reservoir_kernel(
	const int batch_size,
	const int t, const float leak_rate,
	 float ** __restrict__ u_ffwd, const int u_ffwd_rows, const int u_ffwd_cols, const int u_ffwd_stride,
	 float ** __restrict__ u, const int u_rows, const int u_cols, const int u_stride,

	float ** __restrict__ p, const int p_rows, const int p_cols, const int p_stride,
	float ** __restrict__ x_res, const int x_res_rows, const int x_res_cols, const int x_res_stride
)
{
	for (int batch = blockIdx.y * blockDim.y + threadIdx.y; batch < batch_size; batch += gridDim.y * blockDim.y)
	{
		float *P = p[batch];
		float *U = u[batch];
		float *X = x_res[batch];
		float *U_ffwd = u_ffwd[batch];

		float *u_ffwd_t = &U_ffwd[t * u_ffwd_stride];
		assert(t < u_ffwd_rows);
//#pragma unroll 8
		for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < x_res_cols ; col += gridDim.x * blockDim.x)
		{
			float4 _p = reinterpret_cast<float4 *>(P)[col];
			float4 _u = reinterpret_cast<float4 *>(U)[col];
			float4 _u_ffwd = reinterpret_cast<float4 *>(u_ffwd_t)[col];

			_p += leak_rate * ( _u_ffwd + _u - _p);
			reinterpret_cast<float4 *>(X)[col] = tanhf(_p);
			reinterpret_cast<float4 *>(P)[col] = _p;
		}
	}
}
__global__
static void batched_update_reservoir_no_input_kernel(
	const int batch_size,
	const int t, const float leak_rate,
	float ** __restrict__ u, const int u_rows, const int u_cols, const int u_stride,

	float ** __restrict__ p, const int p_rows, const int p_cols, const int p_stride,
	float ** __restrict__ x_res, const int x_res_rows, const int x_res_cols, const int x_res_stride
)
{
	for (int batch = blockIdx.y * blockDim.y + threadIdx.y; batch < batch_size; batch += gridDim.y * blockDim.y)
	{
		float *P = p[batch];
		float *U = u[batch];
		float *X = x_res[batch];

		for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < x_res_cols; col += gridDim.x * blockDim.x)
		{
			float4 _p = reinterpret_cast<float4 *>(P)[col];
			float4 _u = reinterpret_cast<float4 *>(U)[col];

			_p += leak_rate * (_u - _p);
			reinterpret_cast<float4 *>(X)[col] = tanhf(_p);
			reinterpret_cast<float4 *>(P)[col] = _p;
		}
	}
}
__host__
static inline void batched_update_reservoir_no_input
(
	const hipStream_t &stream,
	const std::size_t &batch_size, const std::size_t t, const float &leak_rate, 

	float **u, const std::size_t &u_rows, const std::size_t &u_cols, const std::size_t &u_stride,
	float **p, const std::size_t &p_rows, const std::size_t &p_cols, const std::size_t &p_stride,
	float **x_res, const std::size_t &x_res_rows, const std::size_t &x_res_cols, const std::size_t &x_res_stride
)
{
	dim3 grid, block;

	block.x = 128;
	block.y = 1;

	grid.x = (x_res_cols / 4 + block.x - 1) / block.x;
	grid.y = (batch_size + block.y - 1) / block.y;

	batched_update_reservoir_no_input_kernel << < grid, block, 0, stream >> > (
		batch_size, t, leak_rate,
		
		u, u_rows, u_cols / 4, u_stride,

		p, p_rows, p_cols / 4, p_stride,
		x_res, x_res_rows, x_res_cols / 4, x_res_stride);

	checkCudaErrors(hipGetLastError());
}

__host__
static inline void batched_update_reservoir
(
	const hipStream_t &stream,
	const std::size_t &batch_size, const std::size_t t, const float &leak_rate, 
	 float **u_ffwd, const std::size_t &u_ffwd_rows, const std::size_t &u_ffwd_cols, const std::size_t &u_ffwd_stride,
	 float **u, const std::size_t &u_rows, const std::size_t &u_cols, const std::size_t &u_stride,
	float **p, const std::size_t &p_rows, const std::size_t &p_cols, const std::size_t &p_stride,
	float **x_res, const std::size_t &x_res_rows, const std::size_t &x_res_cols, const std::size_t &x_res_stride
)
{
	dim3 grid, block;

	block.x = 128;
	block.y = 1;

	grid.x = (x_res_cols / 4 + block.x - 1) / block.x;
	grid.y = (batch_size + block.y - 1) / block.y;

	batched_update_reservoir_kernel  << < grid, block, 0, stream >> > (
		batch_size, t, leak_rate,
		u_ffwd, u_ffwd_rows, u_ffwd_cols, u_ffwd_stride,
		u, u_rows, u_cols/4, u_stride,

		p, p_rows, p_cols/4, p_stride,
		x_res, x_res_rows, x_res_cols/4, x_res_stride);

	checkCudaErrors(hipGetLastError());
}
__device__
static const char *dev_cudaGetErrorEnum(hipError_t error)
{
	switch (error)
	{
		case hipSuccess:
			return "hipSuccess";

		case hipErrorMissingConfiguration:
			return "hipErrorMissingConfiguration";

		case hipErrorOutOfMemory:
			return "hipErrorOutOfMemory";

		case hipErrorNotInitialized:
			return "hipErrorNotInitialized";

		case hipErrorLaunchFailure:
			return "hipErrorLaunchFailure";

		case hipErrorPriorLaunchFailure:
			return "hipErrorPriorLaunchFailure";

		case hipErrorLaunchTimeOut:
			return "hipErrorLaunchTimeOut";

		case hipErrorLaunchOutOfResources:
			return "hipErrorLaunchOutOfResources";

		case hipErrorInvalidDeviceFunction:
			return "hipErrorInvalidDeviceFunction";

		case hipErrorInvalidConfiguration:
			return "hipErrorInvalidConfiguration";

		case hipErrorInvalidDevice:
			return "hipErrorInvalidDevice";

		case hipErrorInvalidValue:
			return "hipErrorInvalidValue";

		case hipErrorInvalidPitchValue:
			return "hipErrorInvalidPitchValue";

		case hipErrorInvalidSymbol:
			return "hipErrorInvalidSymbol";

		case hipErrorMapFailed:
			return "hipErrorMapFailed";

		case hipErrorUnmapFailed:
			return "hipErrorUnmapFailed";

		case cudaErrorInvalidHostPointer:
			return "cudaErrorInvalidHostPointer";

		case hipErrorInvalidDevicePointer:
			return "hipErrorInvalidDevicePointer";

		case hipErrorInvalidTexture:
			return "hipErrorInvalidTexture";

		case cudaErrorInvalidTextureBinding:
			return "cudaErrorInvalidTextureBinding";

		case hipErrorInvalidChannelDescriptor:
			return "hipErrorInvalidChannelDescriptor";

		case hipErrorInvalidMemcpyDirection:
			return "hipErrorInvalidMemcpyDirection";

		case cudaErrorAddressOfConstant:
			return "cudaErrorAddressOfConstant";

		case cudaErrorTextureFetchFailed:
			return "cudaErrorTextureFetchFailed";

		case cudaErrorTextureNotBound:
			return "cudaErrorTextureNotBound";

		case cudaErrorSynchronizationError:
			return "cudaErrorSynchronizationError";

		case cudaErrorInvalidFilterSetting:
			return "cudaErrorInvalidFilterSetting";

		case cudaErrorInvalidNormSetting:
			return "cudaErrorInvalidNormSetting";

		case cudaErrorMixedDeviceExecution:
			return "cudaErrorMixedDeviceExecution";

		case hipErrorDeinitialized:
			return "hipErrorDeinitialized";

		case hipErrorUnknown:
			return "hipErrorUnknown";

		case cudaErrorNotYetImplemented:
			return "cudaErrorNotYetImplemented";

		case cudaErrorMemoryValueTooLarge:
			return "cudaErrorMemoryValueTooLarge";

		case hipErrorInvalidHandle:
			return "hipErrorInvalidHandle";

		case hipErrorNotReady:
			return "hipErrorNotReady";

		case hipErrorInsufficientDriver:
			return "hipErrorInsufficientDriver";

		case hipErrorSetOnActiveProcess:
			return "hipErrorSetOnActiveProcess";

		case cudaErrorInvalidSurface:
			return "cudaErrorInvalidSurface";

		case hipErrorNoDevice:
			return "hipErrorNoDevice";

		case hipErrorECCNotCorrectable:
			return "hipErrorECCNotCorrectable";

		case hipErrorSharedObjectSymbolNotFound:
			return "hipErrorSharedObjectSymbolNotFound";

		case hipErrorSharedObjectInitFailed:
			return "hipErrorSharedObjectInitFailed";

		case hipErrorUnsupportedLimit:
			return "hipErrorUnsupportedLimit";

		case cudaErrorDuplicateVariableName:
			return "cudaErrorDuplicateVariableName";

		case cudaErrorDuplicateTextureName:
			return "cudaErrorDuplicateTextureName";

		case cudaErrorDuplicateSurfaceName:
			return "cudaErrorDuplicateSurfaceName";

		case cudaErrorDevicesUnavailable:
			return "cudaErrorDevicesUnavailable";

		case hipErrorInvalidImage:
			return "hipErrorInvalidImage";

		case hipErrorNoBinaryForGpu:
			return "hipErrorNoBinaryForGpu";

		case cudaErrorIncompatibleDriverContext:
			return "cudaErrorIncompatibleDriverContext";

		case hipErrorPeerAccessAlreadyEnabled:
			return "hipErrorPeerAccessAlreadyEnabled";

		case hipErrorPeerAccessNotEnabled:
			return "hipErrorPeerAccessNotEnabled";

		case hipErrorContextAlreadyInUse:
			return "hipErrorContextAlreadyInUse";

		case hipErrorProfilerDisabled:
			return "hipErrorProfilerDisabled";

		case hipErrorProfilerNotInitialized:
			return "hipErrorProfilerNotInitialized";

		case hipErrorProfilerAlreadyStarted:
			return "hipErrorProfilerAlreadyStarted";

		case hipErrorProfilerAlreadyStopped:
			return "hipErrorProfilerAlreadyStopped";

			/* Since CUDA 4.0*/
		case hipErrorAssert:
			return "hipErrorAssert";

		case cudaErrorTooManyPeers:
			return "cudaErrorTooManyPeers";

		case hipErrorHostMemoryAlreadyRegistered:
			return "hipErrorHostMemoryAlreadyRegistered";

		case hipErrorHostMemoryNotRegistered:
			return "hipErrorHostMemoryNotRegistered";

			/* Since CUDA 5.0 */
		case hipErrorOperatingSystem:
			return "hipErrorOperatingSystem";

		case hipErrorPeerAccessUnsupported:
			return "hipErrorPeerAccessUnsupported";

		case cudaErrorLaunchMaxDepthExceeded:
			return "cudaErrorLaunchMaxDepthExceeded";

		case cudaErrorLaunchFileScopedTex:
			return "cudaErrorLaunchFileScopedTex";

		case cudaErrorLaunchFileScopedSurf:
			return "cudaErrorLaunchFileScopedSurf";

		case cudaErrorSyncDepthExceeded:
			return "cudaErrorSyncDepthExceeded";

		case cudaErrorLaunchPendingCountExceeded:
			return "cudaErrorLaunchPendingCountExceeded";

		case cudaErrorNotPermitted:
			return "cudaErrorNotPermitted";

		case hipErrorNotSupported:
			return "hipErrorNotSupported";

			/* Since CUDA 6.0 */
		case cudaErrorHardwareStackError:
			return "cudaErrorHardwareStackError";

		case cudaErrorIllegalInstruction:
			return "cudaErrorIllegalInstruction";

		case cudaErrorMisalignedAddress:
			return "cudaErrorMisalignedAddress";

		case cudaErrorInvalidAddressSpace:
			return "cudaErrorInvalidAddressSpace";

		case cudaErrorInvalidPc:
			return "cudaErrorInvalidPc";

		case hipErrorIllegalAddress:
			return "hipErrorIllegalAddress";

			/* Since CUDA 6.5*/
		case hipErrorInvalidKernelFile:
			return "hipErrorInvalidKernelFile";

		case hipErrorInvalidGraphicsContext:
			return "hipErrorInvalidGraphicsContext";

		case cudaErrorStartupFailure:
			return "cudaErrorStartupFailure";

		case cudaErrorApiFailureBase:
			return "cudaErrorApiFailureBase";

			/* Since CUDA 8.0*/
		case cudaErrorNvlinkUncorrectable:
			return "cudaErrorNvlinkUncorrectable";
	}

	return "<unknown>";
}

template< typename T >
__device__
void dev_check(T result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		printf("CUDA error at %s:%d code=%d\"%s\" \n",
			file, line, static_cast<unsigned int>(result),  func);
			// Make sure we call CUDA Device Reset before exiting
	}
}
#define dev_checkCudaErrors(val)           dev_check ( (val), #val, __FILE__, __LINE__ )
#include <thrust/system_error.h>
#include <thrust/system/cuda/error.h>
#include <sstream>




template <bool gather_states>
static void copy_states(
	const hipStream_t &stream, const std::size_t &batch_size, const std::size_t &t, const std::size_t &ts,
	const std::size_t &stimulus_size,
	const std::size_t &reservoir_size,
	const std::size_t &prediction_size,
	const std::size_t &stimulus_stride,
	const std::size_t &reservoir_stride,
	const std::size_t &prediction_stride,
	const float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	const float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	const float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	const float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float *states, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride)
{}

template <>
static void copy_states<true>(const hipStream_t &stream, const std::size_t &batch_size, const std::size_t &t, const std::size_t &ts,
	const std::size_t &stimulus_size,
	const std::size_t &reservoir_size,
	const std::size_t &prediction_size,
	const std::size_t &stimulus_stride,
	const std::size_t &reservoir_stride,
	const std::size_t &prediction_stride,
	const float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	const float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	const float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	const float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float *states, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride)
{	
	std::vector<float *> incoming_ptr(batch_size);
	std::vector<float *> expected_ptr(batch_size);
	std::vector<float *> x_ro_ptr(batch_size);
	std::vector<float *> x_res_ptr(batch_size);
	/*hipStream_t incoming, expected, x_ro, x_res;*/
	/*hipEvent_t incoming_terminated, expected_terminated, x_ro_terminated, x_res_terminated;
	checkCudaErrors(hipStreamCreateWithFlags(&incoming, hipStreamNonBlocking));
	checkCudaErrors(hipStreamCreateWithFlags(&expected, hipStreamNonBlocking));
	checkCudaErrors(hipStreamCreateWithFlags(&x_ro, hipStreamNonBlocking));
	checkCudaErrors(hipStreamCreateWithFlags(&x_res, hipStreamNonBlocking));*/
	checkCudaErrors(hipMemcpyAsync(incoming_ptr.data(), batched_incoming, batch_size * sizeof(float *), hipMemcpyKind::hipMemcpyDeviceToHost, stream));
	checkCudaErrors(hipMemcpyAsync(expected_ptr.data(), batched_expected, batch_size * sizeof(float *), hipMemcpyKind::hipMemcpyDeviceToHost, stream));
	checkCudaErrors(hipMemcpyAsync(x_ro_ptr.data(), batched_x_ro, batch_size * sizeof(float *), hipMemcpyKind::hipMemcpyDeviceToHost, stream));
	checkCudaErrors(hipMemcpyAsync(x_res_ptr.data(), batched_x_res, batch_size * sizeof(float *), hipMemcpyKind::hipMemcpyDeviceToHost, stream));

	/*checkCudaErrors(hipEventCreate(&incoming_terminated));
	checkCudaErrors(hipEventCreate(&expected_terminated));
	checkCudaErrors(hipEventCreate(&x_ro_terminated));
	checkCudaErrors(hipEventCreate(&x_res_terminated));

	checkCudaErrors(hipStreamSynchronize(incoming));
	checkCudaErrors(hipStreamSynchronize(expected));
	checkCudaErrors(hipStreamSynchronize(x_ro));*/
	checkCudaErrors(hipStreamSynchronize(stream));

	std::size_t offset = 0;
	float *states_ts = &states[ts * states_stride];

	for (std::size_t batch = 0; batch < batch_size; batch++)
	{
		std::size_t offset = 0;
		std::size_t  stimulus_col = batch * stimulus_stride + batch_size * offset;
		checkCudaErrors(hipMemcpyAsync(
			states_ts + stimulus_col,
			&incoming_ptr[batch][t * batched_incoming_strides],
			sizeof(float) * stimulus_size, hipMemcpyKind::hipMemcpyDeviceToDevice, stream));
		offset += stimulus_stride;

		std::size_t  desired_col = batch * prediction_stride + batch_size * offset;
		checkCudaErrors(hipMemcpyAsync(
			states_ts + desired_col,
			&expected_ptr[batch][t * batched_expected_strides],
			sizeof(float) * prediction_size, hipMemcpyKind::hipMemcpyDeviceToDevice, stream));
		offset += prediction_stride;

		std::size_t  reservoir_col = batch * reservoir_stride + batch_size * offset;
		checkCudaErrors(hipMemcpyAsync(
			states_ts + reservoir_col,
			x_res_ptr[batch],
			sizeof(float) * reservoir_size, hipMemcpyKind::hipMemcpyDeviceToDevice, stream));
		offset += reservoir_stride;

		std::size_t  predicted_col = batch * prediction_stride + batch_size * offset;
		checkCudaErrors(hipMemcpyAsync(
			states_ts + predicted_col,
			x_ro_ptr[batch],
			sizeof(float) * prediction_size, hipMemcpyKind::hipMemcpyDeviceToDevice, stream));
		offset += prediction_stride;
	}
	
}


template <bool overwrite_states>
static inline void initialize_states(const hipStream_t &stream,  unsigned long &seed,
	const std::size_t &batch_size,
	float **batched_ptr, const std::size_t &batched_ptr_rows, const std::size_t &batched_ptr_cols, const std::size_t &batched_ptr_stride,
	const float &initial_state_scale)
{
}
template <>
static inline void initialize_states<true>(const hipStream_t &stream, unsigned long &seed,
	const std::size_t &batch_size,
	float **batched_ptr, const std::size_t &batched_ptr_rows, const std::size_t &batched_ptr_cols, const std::size_t &batched_ptr_stride,
	const float &initial_state_scale)
{
	random_uniform(stream, seed, -initial_state_scale, initial_state_scale, 0.0f, batch_size, batched_ptr_rows, batched_ptr_cols, batched_ptr, batched_ptr_stride);
	seed += batch_size * batched_ptr_rows * batched_ptr_cols;
}

static inline void sgemm_nt(
	const hipblasHandle_t handle, const int batch_size,
	const float alpha, const float beta,
	const float **a, const int a_rows, const int a_cols, const int a_stride,
	const float **b, const int b_rows, const int b_cols, const int b_stride,
	float **c, const int c_rows, const int c_cols, const int c_stride
	)
{
	auto op_a_rows = a_rows;
	auto op_a_cols = a_cols;
	auto op_b_rows = b_cols;
	auto op_b_cols = b_rows;

	assert(op_a_rows == c_rows);
	auto m = op_a_rows;
	assert(op_b_cols == c_cols);
	auto n = op_b_cols;
	assert(op_a_cols == op_b_rows);
	auto k = op_a_cols;

	checkCudaErrors(hipblasSgemmBatched(handle,
		hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_T,
		m,n,k,
		&alpha,
		a, a_stride,
		b, b_stride,
		&beta,
		c, c_stride,
		batch_size
	));
}

static inline void sgemm_tn(
	const hipblasHandle_t handle, const int batch_size,
	const float alpha, const float beta,
	const float **a, const int a_rows, const int a_cols, const int a_stride,
	const float **b, const int b_rows, const int b_cols, const int b_stride,
	float **c, const int c_rows, const int c_cols, const int c_stride
)
{
	auto op_a_rows = a_cols;
	auto op_a_cols = a_rows;
	auto op_b_rows = b_rows;
	auto op_b_cols = b_cols;

	assert(op_a_rows == c_rows);
	auto m = op_a_rows;
	assert(op_b_cols == c_cols);
	auto n = op_b_cols;
	assert(op_a_cols == op_b_rows);
	auto k = op_a_cols;

	checkCudaErrors(hipblasSgemmBatched(handle,
		hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_N,
		m, n, k,
		&alpha,
		a, a_stride,
		b, b_stride,
		&beta,
		c, c_stride,
		batch_size
	));
}
static inline void sgemm_nn(
	const hipblasHandle_t handle, const int batch_size,
	const float alpha, const float beta,
	const float **a, const int a_rows, const int a_cols, const int a_stride,
	const float **b, const int b_rows, const int b_cols, const int b_stride,
	float **c, const int c_rows, const int c_cols, const int c_stride
)
{
	auto op_a_rows = a_rows;
	auto op_a_cols = a_cols;
	auto op_b_rows = b_rows;
	auto op_b_cols = b_cols;

	assert(op_a_rows == c_rows);
	auto m = op_a_rows;
	assert(op_b_cols == c_cols);
	auto n = op_b_cols;
	assert(op_a_cols == op_b_rows);
	auto k = op_a_cols;

	checkCudaErrors(hipblasSgemmBatched(handle,
		hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_N,
		m, n, k,
		&alpha,
		a, a_stride,
		b, b_stride,
		&beta,
		c, c_stride,
		batch_size
	));
}

__global__
static void update_readout_error_kernel(
	const int batch_size,
	const int t, 
	const float learning_rate,
	 float ** __restrict__ batched_x_ro, const int batched_x_ro_rows, const int batched_x_ro_cols, const int batched_x_ro_stride,
	 float ** __restrict__ batched_expected, const int batched_expected_rows, const int batched_expected_cols, const int batched_expected_stride,
	float ** __restrict__ batched_error, const int batched_error_rows, const int batched_error_cols, const int batched_error_stride
)
{
	for (int batch = blockIdx.y * blockDim.y + threadIdx.y; batch < batch_size; batch += gridDim.y * blockDim.y)
	{
		float *E = batched_error[batch];
		 float *D = batched_expected[batch];
		 float *X = batched_x_ro[batch];

		for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < batched_error_cols >> 2; col += gridDim.x * blockDim.x)
		{
			float4 d = reinterpret_cast<float4 *>(&D[t * batched_expected_cols])[col];
			float4 x = reinterpret_cast<float4 *>(X)[col];
			float4 e = learning_rate * (d - x);
			/*assert(!isnan(d));
			assert(!isnan(x));
			assert(!isnan(e));*/
			reinterpret_cast<float4 *>(E)[col] = e;
		}
	}
}
#define BLOCK_X 32
#define BLOCK_Y 32


__global__
static void widrow_hoff_kernel(
	const int batch_size, 
	float **__restrict__  batched_w_ro, const int batched_w_ro_rows, const int batched_w_ro_cols, const int batched_w_ro_stride,
	float **__restrict__  batched_x_res, const int batched_x_res_rows, const int batched_x_res_cols, const int batched_x_res_stride,
	float **__restrict__  batched_error, const int batched_error_rows, const int batched_error_cols, const int batched_error_stride
)
{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		float *W = batched_w_ro[batch];
		float *E = batched_error[batch];
		float *X = batched_x_res[batch];

		for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < batched_w_ro_rows; row += gridDim.y * blockDim.y)
		{
			const float e = E[row];
			//assert(!isnan(e));
			for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < batched_w_ro_cols >> 2; col += gridDim.x * blockDim.x)
			{
			
				//assert(!isnan(x));
				reinterpret_cast<float4 *>(&W[row * batched_w_ro_stride])[col] += reinterpret_cast<float4 *>(X)[col] * e;
			}
		}
	}

}
template <typename Parameter>
static inline void update_readout(
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const std::size_t &batch_size, const std::size_t & t, const Parameter &parameter,
	 float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t & batched_x_res_stride,
	 float **batched_x_ro, const std::size_t & batched_x_ro_rows, const std::size_t & batched_x_ro_cols, const std::size_t & batched_x_ro_stride,
	 float **batched_expected, const std::size_t & batched_expected_rows, const std::size_t & batched_expected_cols, const std::size_t &batched_expected_stride,
	float **batched_error, const std::size_t &batched_error_rows, const std::size_t &batched_error_cols, const std::size_t & batched_error_stride,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t & batched_w_ro_stride)
{

}



template <>
static inline void update_readout(
	const hipStream_t &stream, 
	const hipblasHandle_t &handle, 
	const std::size_t &batch_size, const std::size_t & t, const Widrow_Hoff &parameter,
	 float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t & batched_x_res_stride,
	 float **batched_x_ro, const std::size_t & batched_x_ro_rows, const std::size_t & batched_x_ro_cols, const std::size_t & batched_x_ro_stride,
	 float **batched_expected, const std::size_t & batched_expected_rows, const std::size_t & batched_expected_cols, const std::size_t &batched_expected_stride,
	float **batched_error, const std::size_t &batched_error_rows, const std::size_t &batched_error_cols, const std::size_t & batched_error_stride,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t & batched_w_ro_stride)
{
	assert(batched_x_res_rows == 1);
	assert(batched_x_ro_rows == 1);
	assert(t < batched_expected_rows);
	assert(batched_w_ro_cols == batched_x_res_cols);
	assert(batched_w_ro_rows == batched_x_ro_cols);

	{
		dim3 block;
		dim3 grid;

		block.x = warpSize * 4;
		grid.x = (batched_error_cols / 4+ block.x - 1) / block.x;

		block.y = 1;
		grid.y = (batch_size + block.y - 1) / block.y;

		update_readout_error_kernel << <grid, block, 0, stream >> >
			(
				batch_size, t, parameter.get_learning_rate(),
				batched_x_ro, batched_x_ro_rows, batched_x_ro_cols, batched_x_ro_stride,
				batched_expected, batched_expected_rows, batched_expected_cols, batched_expected_stride,
				batched_error, batched_error_rows, batched_error_cols, batched_error_stride
				);

		checkCudaErrors(hipGetLastError());
	}
	

	{
		dim3 block;
		dim3 grid;

		block.x = BLOCK_X;
		block.y = BLOCK_Y;
		block.z = 1;
		grid.x = (batched_w_ro_cols / 4 + block.x - 1) / block.x;
		grid.y = (batched_w_ro_rows + block.y - 1) / block.y;
		grid.z = (batch_size + block.z - 1) / block.z;
		widrow_hoff_kernel << <grid, block, 0, stream >> >
			(
				batch_size,
				batched_w_ro, batched_w_ro_rows, batched_w_ro_cols, batched_w_ro_stride,
				batched_x_res, batched_x_res_rows, batched_x_res_cols, batched_x_res_stride,
				batched_error, batched_error_rows, batched_error_cols, batched_error_stride
				);

		checkCudaErrors(hipGetLastError());
	}
}
static const float one = 1.0f;
static const float zero = 0.0f;

struct isnan_test {
	__host__ __device__ bool operator()(const float a) const {
		return isnan(a);
	}
};


__global__
static void isnan_kernel(
	const int batch_size,
	float **__restrict__  batched_x, const int batched_x_rows, const int batched_x_cols, const int batched_x_stride,
	bool *result
)
{
	*result = false;
	for (int batch = 0; batch < batch_size; batch++)
	{
		float *x = batched_x[batch];
		auto r = thrust::transform_reduce(thrust::seq, x, x + batched_x_rows * batched_x_stride, isnan_test(), false, thrust::plus<bool>());
		*result |= r;

	}
}

void isnan(
	const hipStream_t &stream, 
	const int batch_size,
	float **__restrict__  batched_x, const int batched_x_rows, const int batched_x_cols, const int batched_x_stride)
{
	bool *dev_result = NULL;
	bool result = false;
	checkCudaErrors(hipMalloc((void **)&dev_result, sizeof(bool)));
	isnan_kernel << <1, 1, 0, stream >> > (batch_size, batched_x, batched_x_rows, batched_x_cols, batched_x_stride, dev_result);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpyAsync(&result, dev_result, sizeof(bool), hipMemcpyKind::hipMemcpyDeviceToHost, stream));
	checkCudaErrors(hipStreamSynchronize(stream));
	if (result)
		throw std::runtime_error("NAN");
}

template<bool gather_states, bool overwrite_states, typename Parameter>
void update_model(
	const std::size_t &batch_size,
	unsigned long &seed,
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const Parameter &parameter,
	const std::size_t &stimulus_stride, const std::size_t &reservoir_stride, const std::size_t &prediction_stride,
	const std::size_t &stimulus_size, const std::size_t &reservoir_size, const std::size_t &prediction_size,
	const float &leak_rate, const float &initial_state_scale,
	float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	float **batched_w_ffwd, const std::size_t &batched_w_ffwd_rows, const std::size_t &batched_w_ffwd_cols, const std::size_t &batched_w_ffwd_strides,
	float **batched_u_ffwd, const std::size_t &batched_u_ffwd_rows, const std::size_t &batched_u_ffwd_cols, const std::size_t &batched_u_ffwd_strides,
	float **batched_x_in, const std::size_t &batched_x_in_rows, const std::size_t &batched_x_in_cols, const std::size_t &batched_x_in_strides,
	float **batched_w_in, const std::size_t &batched_w_in_rows, const std::size_t &batched_w_in_cols, const std::size_t &batched_w_in_strides,
	float **batched_u, const std::size_t &batched_u_rows, const std::size_t &batched_u_cols, const std::size_t &batched_u_strides,
	float **batched_p, const std::size_t &batched_p_rows, const std::size_t &batched_p_cols, const std::size_t &batched_p_strides,
	float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t &batched_w_ro_strides,
	float **batched_error, const std::size_t & batched_error_rows, const std::size_t &batched_error_cols, const std::size_t &batched_error_strides,
	const int *offsets, const int *durations, const std::size_t &repetitions,
	float *states_samples, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride
	)
{
	/*isnan(stream, batch_size, batched_w_ffwd, batched_w_ffwd_rows, batched_w_ffwd_cols, batched_w_ffwd_strides);
	isnan(stream, batch_size, batched_incoming, batched_incoming_rows, batched_incoming_cols, batched_incoming_strides);*/
	sgemm_tn(
		handle, batch_size,
		one, zero,
		(const float **)batched_w_ffwd, batched_w_ffwd_cols, batched_w_ffwd_rows, batched_w_ffwd_strides,
		(const float **)batched_incoming, batched_incoming_cols, batched_incoming_rows, batched_incoming_strides,
	

		batched_u_ffwd, batched_u_ffwd_cols, batched_u_ffwd_rows, batched_u_ffwd_strides
	);




	std::size_t ts = 0;
	for (std::size_t repetition = 0; repetition < repetitions; repetition++)
	{
		initialize_states<overwrite_states>(stream,  seed, batch_size,
			batched_p, batched_p_rows, batched_p_cols, batched_p_strides, initial_state_scale);
		//isnan(stream, batch_size, batched_p, batched_p_rows, batched_p_cols, batched_p_strides);
		initialize_states<overwrite_states>(stream, seed, batch_size,
			(float **)batched_x_in, batched_x_in_rows, batched_x_in_cols, batched_x_in_strides, initial_state_scale);
		//isnan(stream, batch_size, batched_x_in, batched_x_in_rows, batched_x_in_cols, batched_x_in_strides);
		for (std::size_t k = 0; k < durations[repetition]; k++, ts++)
		{
			int t = offsets[ts];
	
			//std::cout << "t = " << t << std::endl;
			batched_reset(stream, batch_size, batched_u_rows, batched_u_cols, batched_u, batched_u_strides);
			//isnan(stream, batch_size, batched_u, batched_u_rows, batched_u_cols, batched_u_strides);
			batched_sgemv(stream, batch_size,
				batched_w_in, batched_w_in_rows, batched_w_in_cols, batched_w_in_strides,
				batched_x_in, batched_x_in_rows, batched_x_in_cols, batched_x_in_strides,
				batched_u, batched_u_rows, batched_u_cols, batched_u_strides
			);
			//isnan(stream, batch_size, batched_u, batched_u_rows, batched_u_cols, batched_u_strides);

			if (t < 0)
			{
				t = -t;
				batched_update_reservoir_no_input(
					stream,
					batch_size, t, leak_rate,
					batched_u, batched_u_rows, batched_u_cols, batched_u_strides,
					batched_p, batched_p_rows, batched_p_cols, batched_p_strides,
					batched_x_res, batched_x_res_rows, batched_x_res_cols, batched_x_res_strides);
			}
			else
			{
				batched_update_reservoir(
					stream,
					batch_size, t, leak_rate,
					batched_u_ffwd, batched_u_ffwd_rows, batched_u_ffwd_cols, batched_u_ffwd_strides,
					batched_u, batched_u_rows, batched_u_cols, batched_u_strides,
					batched_p, batched_p_rows, batched_p_cols, batched_p_strides,
					batched_x_res, batched_x_res_rows, batched_x_res_cols, batched_x_res_strides);
			}


			//isnan(stream, batch_size, batched_p, batched_p_rows, batched_p_cols, batched_w_ffwd_strides);
			//isnan(stream, batch_size, batched_x_res, batched_x_res_rows, batched_x_res_cols, batched_x_res_strides);

			batched_reset(stream, batch_size, batched_x_ro_rows, batched_x_ro_cols, batched_x_ro, batched_x_ro_strides);
			//isnan(stream, batch_size, batched_x_ro, batched_x_ro_rows, batched_x_ro_cols, batched_x_ro_strides);
			batched_sgemv(stream, batch_size,
				batched_w_ro, batched_w_ro_rows, batched_w_ro_cols, batched_w_ro_strides,
				batched_x_res, batched_x_res_rows, batched_x_res_cols, batched_x_res_strides,
				batched_x_ro, batched_x_ro_rows, batched_x_ro_cols, batched_x_ro_strides
			);
			//isnan(stream, batch_size, batched_x_ro, batched_x_ro_rows, batched_x_ro_cols, batched_x_ro_strides);
			update_readout<Parameter>(
				stream, handle, batch_size, t, parameter,
				batched_x_res, batched_x_res_rows, batched_x_res_cols, batched_x_res_strides,
				batched_x_ro, batched_x_ro_rows, batched_x_ro_cols, batched_x_ro_strides,
				batched_expected, batched_expected_rows, batched_expected_cols, batched_expected_strides,
				batched_error, batched_error_rows, batched_error_cols, batched_error_strides,
				batched_w_ro, batched_w_ro_rows, batched_w_ro_cols, batched_w_ro_strides);
			//isnan(stream, batch_size, batched_error, batched_error_rows, batched_error_cols, batched_error_strides);
			//isnan(stream, batch_size, batched_w_ro, batched_w_ro_rows, batched_w_ro_cols, batched_w_ro_strides);
			copy_states<gather_states>(stream, batch_size, t, ts,
				stimulus_size, reservoir_size, prediction_size,
				stimulus_stride, reservoir_stride, prediction_stride,
				(const float **)batched_incoming, batched_incoming_rows, batched_incoming_cols, batched_incoming_strides,
				(const float **)batched_expected, batched_expected_rows, batched_expected_cols, batched_expected_strides,
				(const float **)batched_x_ro, batched_x_ro_rows, batched_x_ro_cols, batched_x_ro_strides,
				(const float **)batched_x_res, batched_x_res_rows, batched_x_res_cols, batched_x_res_strides,
				states_samples, states_rows, states_cols, states_stride
				);
		}
	}
}
__global__
static void sum_inplace_kernel(
	const int batch_size, const int place_cells_number, const int size,
	float   *** __restrict__ batched_hypothesis)
{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		for (int place_cell = blockIdx.y * blockDim.y + threadIdx.y; place_cell < place_cells_number; place_cell += gridDim.y * blockDim.y)
		{
			const int place_cell_a = place_cell;
			const int place_cell_b = place_cell + place_cells_number;
			float *hypothesis_a = batched_hypothesis[batch][place_cell_a];
			float *hypothesis_b = batched_hypothesis[batch][place_cell_b];
			for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += gridDim.x * blockDim.x)
			{
				reinterpret_cast<float4 *>(hypothesis_a)[idx] += reinterpret_cast<float4 *>(hypothesis_b)[idx];
			}
		}
	}
}
__global__
static void sum_kernel(
	const int batch_size, const int place_cells_number, const int size,
	float   *** __restrict__ batched_hypothesis,
	float   ** __restrict__ batched_location)
{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		float *location = batched_location[batch];
		for (int place_cell = blockIdx.y * blockDim.y + threadIdx.y; place_cell < place_cells_number; place_cell += gridDim.y * blockDim.y)
		{
			const int place_cell_a = place_cell;
			const int place_cell_b = place_cell + place_cells_number;
			float *hypothesis_a = batched_hypothesis[batch][place_cell_a];
			float *hypothesis_b = batched_hypothesis[batch][place_cell_b];
		
			for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += gridDim.x * blockDim.x)
			{
				reinterpret_cast<float4 *>(location)[idx] = reinterpret_cast<float4 *>(hypothesis_a)[idx] + reinterpret_cast<float4 *>(hypothesis_b)[idx];
			}
		}
	}
}
__global__
static void weighted_acc_inplace_kernel(
	const int batch_size, const int place_cells_number, const int size,
	float *** __restrict__ batched_hypothesis,
	float ** __restrict__ batched_scale,
	float ** __restrict__ batched_location_probability)
{
	assert(place_cells_number == 1);
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		float *location_probability = batched_location_probability[batch];
		for (int place_cell = blockIdx.y * blockDim.y + threadIdx.y; place_cell < place_cells_number; place_cell += gridDim.y * blockDim.y)
		{
			const float scale = batched_scale[batch][place_cell];
			float *hypothesis = batched_hypothesis[batch][place_cell];
			for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += gridDim.x * blockDim.x)
			{
			
				reinterpret_cast<float4 *>(location_probability)[idx] += reinterpret_cast<float4 *>(hypothesis)[idx] / scale;
					
			}
		}
	}
}
__global__
static void weighted_sum_inplace_kernel( 
	const int batch_size,
	const int place_cells_number, const int size,
	float   ** __restrict__ scale, const int scale_stride,
	float   ***__restrict__ batched_hypothesis, const int hypothesis_stride
	)
{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		for (int place_cell = blockIdx.y * blockDim.y + threadIdx.y; place_cell < place_cells_number; place_cell += gridDim.y * blockDim.y)
		{
			const int place_cell_a = place_cell;
			const int place_cell_b = place_cell + place_cells_number;
			float *hypothesis_a = batched_hypothesis[batch][place_cell_a];
			float *hypothesis_b = batched_hypothesis[batch][place_cell_b];
			const float scale_a = scale[batch][place_cell_a];
			const float scale_b = scale[batch][place_cell_b];
			for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += gridDim.x * blockDim.x)
			{
				reinterpret_cast<float4 *>(hypothesis_a)[idx] =
					reinterpret_cast<float4 *>(hypothesis_a)[idx] / scale_a +
					reinterpret_cast<float4 *>(hypothesis_b)[idx] / scale_b;
			}
		}
	}
}


__global__
static void location_hypothesis_kernel(
	const int batch_size,
	const int place_cells_number, const int size, const float minus_inv_sigma2,
	 const float   ** __restrict__ batched_firing_rate_map, const int firing_rate_stride,
	 const float  ** __restrict__ batched_prediction, const int prediction_stride,
	float  *** __restrict__ batched_hypothesis_map, const int hypothesis_stride,
	float  ** __restrict__ batched_scale, const int scale_stride)
{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		float *scale = batched_scale[batch];
		const float *prediction = batched_prediction[batch];
	

		for (int place_cell = blockIdx.y * blockDim.y + threadIdx.y; place_cell < place_cells_number; place_cell += gridDim.y * blockDim.y)
		{
			const float *firing_rate_map = batched_firing_rate_map[place_cell];
			float *hypothesis_map = batched_hypothesis_map[batch][place_cell];
			const float p = prediction[place_cell];

			float4 sum4 = make_float4(0.0f);
			for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += gridDim.x * blockDim.x)
			{
				const float4 value = reinterpret_cast<float4 *>(const_cast<float *>(firing_rate_map))[idx] - p;
				const float4 response = expf(value * value * minus_inv_sigma2);
				sum4 += response;
				reinterpret_cast<float4 *>(hypothesis_map)[idx] = response;
			}

			float sum = sum4.x + sum4.y + sum4.z + sum4.w;
			sum = warpReduceSum(sum);
			if ((threadIdx.x & 31) == 0)
			{
				atomicAdd(&scale[place_cell], sum * place_cells_number);
			}
		}
	
	}
}






void compute_place_cell_location_probability(
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const std::size_t &batch_size, const std::size_t &place_cells_number, const std::size_t &rows, const std::size_t &cols,
	const float &sigma,
	const float ** firing_rate_map, const std::size_t &firing_rate_map_rows, const std::size_t &firing_rate_map_cols, const std::size_t &firing_rate_map_stride,
	float **scale, const std::size_t &scale_rows, const std::size_t &scale_cols, const std::size_t &scale_stride,
	const float **prediction, const std::size_t &prediction_rows, const std::size_t &prediction_cols, const std::size_t &prediction_stride,
	float *** hypothesis_map, const std::size_t &hypothesis_map_rows, const std::size_t &hypothesis_map_cols, const std::size_t &hypothesis_map_stride,
	float ** location_probability, const std::size_t &location_probability_rows, const std::size_t &location_probability_cols, const std::size_t &location_probability_stride) // batch_size * 2

{
	
	const auto minus_sigma2_inv = -1.0/(sigma * sigma);
	const std::size_t size = rows * firing_rate_map_stride;
	dim3 block, grid;
	block.x = 32;
	block.y = 32;
	block.z = 1;
	batched_reset(stream, batch_size, scale_rows, scale_cols, scale, scale_stride);
	grid.x = (size / 4 + block.x - 1) / block.x;
	grid.z = (batch_size + block.z - 1) / block.z;
	{
		grid.y = (place_cells_number + block.y - 1) / block.y;
		// reset location hyppothesis
		location_hypothesis_kernel << <grid, block, 0, stream >> > (
				batch_size, place_cells_number, size / 4, minus_sigma2_inv,
				firing_rate_map, firing_rate_map_stride,
				prediction, prediction_stride,
				hypothesis_map, hypothesis_map_stride,
				scale, scale_stride);
		checkCudaErrors(hipGetLastError());
	}

	std::size_t place_cells_number_range = place_cells_number / 2;
	const std::size_t place_cells_number_remaining = place_cells_number - place_cells_number_range * 2;

	{
		grid.y = (place_cells_number_range + block.y - 1) / block.y;

		weighted_sum_inplace_kernel << <grid, block, 0, stream >> > (
			batch_size, place_cells_number_range, size/4,
			scale, scale_stride,
			hypothesis_map, hypothesis_map_stride);
		checkCudaErrors(hipGetLastError());
	}

	while (place_cells_number_range >= 2)
	{
		place_cells_number_range /= 2;

		grid.y = (place_cells_number_range + block.y - 1) / block.y;

		sum_inplace_kernel << <grid, block, 0, stream >> > (
			batch_size, place_cells_number_range, size/4,
			hypothesis_map);
		checkCudaErrors(hipGetLastError());
	}

	assert(place_cells_number_range == 1);
	{
		grid.y = (place_cells_number_range + block.y - 1) / block.y;

		sum_kernel << <grid, block, 0, stream >> > (
			batch_size, place_cells_number_range, size/4,
			hypothesis_map, 
			location_probability);
		checkCudaErrors(hipGetLastError());
	}


	{
		if (place_cells_number_remaining > 0)
		{
			grid.y = (place_cells_number_range + block.y - 1) / block.y;

			weighted_acc_inplace_kernel << <grid, block, 0, stream >> > (
				batch_size, place_cells_number_range, size/4,
				hypothesis_map,
				scale,
				location_probability);
			checkCudaErrors(hipGetLastError());
		}
	}
}

__global__
static void inside_circle_kernel(
	const int batch_size, const int rows, const int cols, const int location_probability_stride,
	const float radius2,
	const float scale,
	const float   * __restrict__ x_grid,
	const float   * __restrict__ y_grid,
	const float ** __restrict__ batched_current_location,
	float   ** __restrict__ batched_location_probability)

{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		float *location_probability = batched_location_probability[batch];
		const float *current_location = batched_current_location[batch];
		const float x = current_location[0];
		const float y = current_location[1];
		for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < rows; row += gridDim.y * blockDim.y)
		{
			const float dy = y_grid[row] - y;
			const float dy2 = dy * dy;
			for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < cols; col += gridDim.x * blockDim.x)
			{
				const float4 dx = reinterpret_cast<float4 *>(const_cast<float *>(x_grid))[col] - x;
				const float4 dx2 = dx * dx;
				const float4 lhs = dx2 + dy2;
				float4 p = reinterpret_cast<float4 *>(&location_probability[row * location_probability_stride])[col];

				hiprandStatePhilox4_32_10_t state;
		
				// seed a random number generator
				hiprand_init(col * rows + row + batch * rows * cols, 0, 0, &state);

				p += hiprand_uniform4(&state) * scale;
				float4 s;
				s.x = lhs.x > radius2 ? 0.0f : p.x;
				s.y = lhs.y > radius2 ? 0.0f : p.y;
				s.z = lhs.z > radius2 ? 0.0f : p.z;
				s.w = lhs.w > radius2 ? 0.0f : p.w;

				reinterpret_cast<float4 *>(&location_probability[row * location_probability_stride])[col] = s;
			}
		}
	
	}
}



void compute_reachable_locations(
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const std::size_t &batch_size, const std::size_t &place_cells_number, const std::size_t &rows, const std::size_t &cols,
	const float &radius, const float &scale,
	const float *x_grid, const std::size_t &x_grid_rows, const std::size_t &x_grid_cols, const std::size_t &x_grid_stride,
	const float *y_grid, const std::size_t &y_grid_rows, const std::size_t &y_grid_cols, const std::size_t &y_grid_stride,
	const float **batched_current_location, const std::size_t &batched_current_location_rows, const std::size_t &batched_current_location_cols, const std::size_t &batched_current_location_stride,
	float **batched_x_grid_centered2, const std::size_t &batched_x_grid_centered2_rows, const std::size_t &batched_x_grid_centered2_cols, const std::size_t &batched_x_grid_centered2_stride,
	float **batched_y_grid_centered2, const std::size_t &batched_y_grid_centered2_rows, const std::size_t &batched_y_grid_centered2_cols, const std::size_t &batched_y_grid_centered2_stride,
	float  **batched_location_probability, const std::size_t &batched_location_probability_rows, const std::size_t &batched_location_probability_cols, const std::size_t &batched_location_probability_strides)
{

	dim3 grid, block;
	block.x = BLOCK_X;
	block.y = BLOCK_Y;
	block.z = 1;

	grid.x = (batched_location_probability_cols / 4 + block.x - 1) / block.x;
	grid.y = (batched_location_probability_rows  + block.y - 1) / block.y;
	grid.z = (batch_size + block.z - 1) / block.z;

		inside_circle_kernel << <grid, block, 0, stream >> > (batch_size, rows, cols / 4, batched_location_probability_strides, radius * radius, scale,
			x_grid, y_grid, batched_current_location, batched_location_probability);
		checkCudaErrors(hipGetLastError());


}




struct normalize_functor
{
	const float scale;
	const float offset;
	__device__
	normalize_functor(float a, float b) : scale(1.0f/ (b - a)) , offset(-a)
	{}
	__device__
	float operator()(const float& x) const
	{ 
		return (x + offset ) * scale; 
	} 
};





__global__
void select_location_kernel(const int batch_size, 
	const float * __restrict__ x_grid, const int x_grid_rows, const int x_grid_cols, const int x_grid_stride,
	const float *__restrict__ y_grid, const int y_grid_rows, const int y_grid_cols, const int y_grid_stride,
	float **__restrict__ batched_location_probability, const int batched_location_probability_rows, const int batched_location_probability_cols, const int batched_location_probability_stride,
	float ** __restrict__ batched_predicted_location, const int batched_predicted_location_rows, const int batched_predicted_location_cols, const int batched_predicted_location_stride

)
{
#ifndef __DEBUG
	for (int batch = blockIdx.x * blockDim.x + threadIdx.x; batch < batch_size; batch += gridDim.x * blockDim.x)
	{


		float *predicted_position = batched_predicted_location[batch];
		float *location_probability = batched_location_probability[batch];
		int idx = -1;
		{
			hipblasHandle_t handle;
			dev_checkCudaErrors(hipblasCreate(&handle));
			dev_checkCudaErrors(hipblasIsamax(handle, batched_location_probability_stride * batched_location_probability_rows, location_probability, 1, &idx));
			dev_checkCudaErrors(hipblasDestroy(handle));
		}

		int row = idx / batched_location_probability_stride;
		int col = idx % batched_location_probability_stride;
		predicted_position[0] = x_grid[col];
		predicted_position[1] = y_grid[row];
	}
#endif
}



void compute_select_most_probable_location(const hipStream_t &stream,  const std::size_t &batch_size, const std::size_t &rows, const std::size_t &cols,
	const float *x_grid, const std::size_t &x_grid_rows, const std::size_t &x_grid_cols, const std::size_t &x_grid_stride,
	const float *y_grid, const std::size_t &y_grid_rows, const std::size_t &y_grid_cols, const std::size_t &y_grid_stride,
	const float  **batched_location_probability, const std::size_t &batched_location_probability_rows, const std::size_t &batched_location_probability_cols, const std::size_t &batched_location_probability_strides,
	float **batched_predicted_location, const std::size_t &batched_predicted_location_rows, const std::size_t &batched_predicted_location_cols, const std::size_t &batched_predicted_location_strides
)
{
	std::vector<float *> batched_location_probability_ptr(batch_size);
	std::vector<float *> batched_predicted_location_ptr(batch_size);
	std::vector<hipcub::KeyValuePair <int, float> *> argmax_ptr(batch_size);
	std::vector<hipcub::KeyValuePair <int, float>> argmax_host(batch_size);
	std::vector<void *> temp_storage_ptr(batch_size);
	std::vector<std::size_t> temp_storage_bytes(batch_size);
	std::vector<hipStream_t> child_streams(batch_size);



	checkCudaErrors(hipMemcpyAsync(batched_location_probability_ptr.data(), batched_location_probability, batch_size * sizeof(float *), hipMemcpyKind::hipMemcpyDeviceToHost, stream));
	checkCudaErrors(hipMemcpyAsync(batched_predicted_location_ptr.data(), batched_predicted_location, batch_size * sizeof(float *), hipMemcpyKind::hipMemcpyDeviceToHost, stream));



	for (int batch = 0; batch < batch_size; batch++)
	{
		float *d_in = batched_location_probability_ptr[batch];
		checkCudaErrors(hipStreamCreateWithFlags(&child_streams[batch], hipStreamNonBlocking));

		checkCudaErrors(hipMalloc(&argmax_ptr[batch], sizeof(hipcub::KeyValuePair <int, float>)));
		hipcub::DeviceReduce::ArgMax(temp_storage_ptr[batch], temp_storage_bytes[batch], d_in, argmax_ptr[batch], batched_location_probability_strides * batched_location_probability_rows, child_streams[batch]);
		// Allocate temporary storage
		checkCudaErrors(hipMalloc(&temp_storage_ptr[batch], temp_storage_bytes[batch]));
		hipcub::DeviceReduce::ArgMax(temp_storage_ptr[batch], temp_storage_bytes[batch], d_in, argmax_ptr[batch], batched_location_probability_strides * batched_location_probability_rows, child_streams[batch]);
		checkCudaErrors(hipMemcpyAsync(&argmax_host[batch], argmax_ptr[batch], sizeof(hipcub::KeyValuePair <int, float>), hipMemcpyKind::hipMemcpyDeviceToHost, child_streams[batch]));
	}


	for (int batch = 0; batch < batch_size; batch++)
	{
		
		checkCudaErrors(hipStreamSynchronize(child_streams[batch]));
		checkCudaErrors(hipStreamDestroy(child_streams[batch]));
		checkCudaErrors(hipFree(temp_storage_ptr[batch]));
		checkCudaErrors(hipFree(argmax_ptr[batch]));

		auto idx = argmax_host[batch].key;
		auto row = idx / batched_location_probability_strides;
		auto col = idx % batched_location_probability_strides;
		checkCudaErrors(hipMemcpyAsync(batched_predicted_location_ptr[batch] + 0, &x_grid[col], sizeof(float), hipMemcpyKind::hipMemcpyDeviceToDevice, stream));
		checkCudaErrors(hipMemcpyAsync(batched_predicted_location_ptr[batch] + 1, &y_grid[row], sizeof(float), hipMemcpyKind::hipMemcpyDeviceToDevice, stream));
		
	}
}


__global__
void reduce_cols_kernel(const int batch_size,
	float **__restrict__ batched_location_probability, const int batched_location_probability_rows, const int batched_location_probability_cols, const int batched_location_probability_stride,
	float **__restrict__ batched_reduced_location_probability, const int batched_reduced_location_probability_rows, const int batched_reduced_location_probability_cols, const int batched_reduced_location_probability_stride
)
{
	for (int batch = blockIdx.z * blockDim.z + threadIdx.z; batch < batch_size; batch += gridDim.z * blockDim.z)
	{
		float *location_proabability = batched_location_probability[batch];
		float *reduced_location_probability = batched_reduced_location_probability[batch];

		for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < batched_location_probability_rows; row += gridDim.y * blockDim.y)
		{
			float sum = 0.0f;
			for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < batched_location_probability_cols >> 2; col += gridDim.x * blockDim.x)
			{
				float4 x = reinterpret_cast<float4 *>(&location_proabability[row * batched_location_probability_stride])[col];
				sum += x.x + x.y + x.z + x.w;
			}
			sum = warpReduceSum(sum);
			if ((threadIdx.x & 31) == 0)
			{
				atomicAdd(&reduced_location_probability[row], sum);
			}
		}
	}
}

static inline void reduce_cols(const hipStream_t &stream, const std::size_t &batch_size,
	const float  **batched_location_probability, const std::size_t &batched_location_probability_rows, const std::size_t &batched_location_probability_cols, const std::size_t &batched_location_probability_strides,
	float **batched_reduced_location_probability, const std::size_t &batched_reduced_location_probability_rows, const std::size_t &batched_reduced_location_probability_cols, const std::size_t &batched_reduced_location_probability_strides
)
{
	dim3 grid, block;
	block.x = BLOCK_X;
	block.y = BLOCK_Y;
	block.z = 1;

	grid.x = (batched_location_probability_cols / 4 + block.x - 1) / block.x;
	grid.y = (batched_location_probability_rows + block.y - 1) / block.y;
	grid.z = (batch_size + block.z - 1) / block.z;
	reduce_cols_kernel << <grid, block, 0, stream >> > (batch_size,
		(float  **)batched_location_probability, batched_location_probability_rows, batched_location_probability_cols, batched_location_probability_strides,
		batched_reduced_location_probability, batched_reduced_location_probability_rows, batched_reduced_location_probability_cols, batched_reduced_location_probability_strides);
	checkCudaErrors(hipGetLastError());
}

__global__
void draw_location_kernel(const int batch_size, const unsigned long seed,
	const float * __restrict__ x_grid, const int x_grid_rows, const int x_grid_cols, const int x_grid_stride,
	const float *__restrict__ y_grid, const int y_grid_rows, const int y_grid_cols, const int y_grid_stride,
	float **__restrict__ batched_location_probability, const int batched_location_probability_rows, const int batched_location_probability_cols, const int batched_location_probability_stride,
	float **__restrict__ batched_reduced_location_probability, const int batched_reduced_location_probability_rows, const int batched_reduced_location_probability_cols, const int batched_reduced_location_probability_stride,
	float **__restrict__ batched_row_cumsum, const int batched_row_cumsum_rows, const int batched_row_cumsum_cols, const int batched_row_cumsum_stride,
	float **__restrict__ batched_col_cumsum, const int batched_col_cumsum_rows, const int batched_col_cumsum_cols, const int batched_col_cumsum_stride,
	float ** __restrict__ batched_predicted_location, const int batched_predicted_location_rows, const int batched_predicted_location_cols, const int batched_predicted_location_stride

)
{
#ifndef __DEBUG
	hiprandState localState;
	hiprand_init(seed, 0, 0, &localState);

	for (int batch = 0; batch < batch_size; batch++)
	{
		float *location_probability = batched_location_probability[batch];
		float *reduced_location_probability = batched_reduced_location_probability[batch];
		float *row_cumsum = batched_row_cumsum[batch];
		float *col_cumsum = batched_col_cumsum[batch];
		float *predicted_position = batched_predicted_location[batch];

		thrust::inclusive_scan(thrust::seq, reduced_location_probability, reduced_location_probability + batched_location_probability_rows, row_cumsum);

		const float row_a = row_cumsum[0];
		const float row_b = row_cumsum[batched_location_probability_rows - 1];
		//printf("before %f %f\n", row_cumsum[0], row_cumsum[batched_location_probability_rows - 1]);
		thrust::transform(thrust::seq, row_cumsum, row_cumsum + batched_location_probability_rows, row_cumsum, normalize_functor(row_a, row_b));
		//printf("after %f %f\n", row_cumsum[0], row_cumsum[batched_location_probability_rows - 1]);
		float ry = hiprand_uniform(&localState);

		int row = thrust::distance(row_cumsum, thrust::lower_bound(thrust::seq, row_cumsum, row_cumsum + batched_location_probability_rows, ry));
		//printf("ry = %f -> row =  %d\n", ry, row);
		assert(0 <= row && row < batched_location_probability_rows);

		float *location_probability_row = &location_probability[row * batched_location_probability_stride];
		thrust::inclusive_scan(thrust::seq, location_probability_row, location_probability_row + batched_location_probability_cols, col_cumsum);
		const float col_a = col_cumsum[0];
		const float col_b = col_cumsum[batched_location_probability_cols - 1];
		//printf("before %f %f\n", col_cumsum[0], col_cumsum[batched_location_probability_cols- 1]);
		thrust::transform(thrust::seq, col_cumsum, col_cumsum + batched_location_probability_cols, col_cumsum, normalize_functor(col_a, col_b));
		//printf("after %f %f\n", col_cumsum[0], col_cumsum[batched_location_probability_cols - 1]);
		float rx = hiprand_uniform(&localState);

		int col = thrust::distance(col_cumsum, thrust::lower_bound(thrust::seq, col_cumsum, col_cumsum + batched_location_probability_cols, rx));
		// printf("rx = %f -> col = %d\n", rx, col);
		assert(0 <= col && col < batched_location_probability_cols);

		predicted_position[0] = x_grid[col];
		predicted_position[1] = y_grid[row];
	}
#endif
}

void compute_draw_probable_location(
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const std::size_t &batch_size, const std::size_t &rows, const std::size_t &cols,
	const float *x_grid, const std::size_t &x_grid_rows, const std::size_t &x_grid_cols, const std::size_t &x_grid_stride,
	const float *y_grid, const std::size_t &y_grid_rows, const std::size_t &y_grid_cols, const std::size_t &y_grid_stride,
	const float  **batched_location_probability, const std::size_t &batched_location_probability_rows, const std::size_t &batched_location_probability_cols, const std::size_t &batched_location_probability_strides,
	float **batched_reduced_location_probability, const std::size_t &batched_reduced_location_probability_rows, const std::size_t &batched_reduced_location_probability_cols, const std::size_t &batched_reduced_location_probability_strides,
	float **batched_row_cumsum, const std::size_t &batched_row_cumsum_rows, const std::size_t &batched_row_cumsum_cols, const std::size_t &batched_row_cumsum_strides,
	float **batched_col_cumsum, const std::size_t &batched_col_cumsum_rows, const std::size_t &batched_col_cumsum_cols, const std::size_t &batched_col_cumsum_strides,
	float **batched_predicted_location, const std::size_t &batched_predicted_location_rows, const std::size_t &batched_predicted_location_cols, const std::size_t &batched_predicted_location_stride
)
{
	batched_reset(stream, batch_size, batched_reduced_location_probability_rows, batched_reduced_location_probability_cols,
		batched_reduced_location_probability, batched_reduced_location_probability_strides);

	reduce_cols(stream, batch_size,
		batched_location_probability, batched_location_probability_rows, batched_location_probability_cols, batched_location_probability_strides,
		batched_reduced_location_probability, batched_reduced_location_probability_rows, batched_reduced_location_probability_cols, batched_reduced_location_probability_strides);

	static unsigned long seed = 0;
	draw_location_kernel << <1, 1, 0, stream >> > (batch_size, seed,
		x_grid, x_grid_rows, x_grid_cols, x_grid_stride,
		y_grid, y_grid_rows, y_grid_cols, y_grid_stride,
		(float  **)batched_location_probability, batched_location_probability_rows, batched_location_probability_cols, batched_location_probability_strides,
		batched_reduced_location_probability, batched_reduced_location_probability_rows, batched_reduced_location_probability_cols, batched_reduced_location_probability_strides,
		batched_row_cumsum, batched_row_cumsum_rows, batched_row_cumsum_cols, batched_row_cumsum_strides,
		batched_col_cumsum, batched_col_cumsum_rows, batched_col_cumsum_cols, batched_col_cumsum_strides,
		batched_predicted_location, batched_predicted_location_rows, batched_predicted_location_cols, batched_predicted_location_stride
		);
	checkCudaErrors(hipGetLastError());
	seed += batch_size * 2;
}



template  void update_model<true, true, Widrow_Hoff>(
	const std::size_t &batch_size,
	unsigned long &seed,
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const Widrow_Hoff &parameter,
	const std::size_t &stimulus_stride, const std::size_t &reservoir_stride, const std::size_t &prediction_stride,
	const std::size_t &stimulus_size, const std::size_t &reservoir_size, const std::size_t &prediction_size,
	const float &leak_rate, const float &initial_state_scale,
	float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	float **batched_w_ffwd, const std::size_t &batched_w_ffwd_rows, const std::size_t &batched_w_ffwd_cols, const std::size_t &batched_w_ffwd_strides,
	float **batched_u_ffwd, const std::size_t &batched_u_ffwd_rows, const std::size_t &batched_u_ffwd_cols, const std::size_t &batched_u_ffwd_strides,
	float **batched_x_in, const std::size_t &batched_x_in_rows, const std::size_t &batched_x_in_cols, const std::size_t &batched_x_in_strides,
	float **batched_w_in, const std::size_t &batched_w_in_rows, const std::size_t &batched_w_in_cols, const std::size_t &batched_w_in_strides,
	float **batched_u, const std::size_t &batched_u_rows, const std::size_t &batched_u_cols, const std::size_t &batched_u_strides,
	float **batched_p, const std::size_t &batched_p_rows, const std::size_t &batched_p_cols, const std::size_t &batched_p_strides,
	float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t &batched_w_ro_strides,
	float **batched_error, const std::size_t & batched_error_rows, const std::size_t &batched_error_cols, const std::size_t &batched_error_strides,
	const int *offsets, const int *durations, const std::size_t &repetitions,
	float *states_samples, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride
	);
template void update_model< true, false, Widrow_Hoff >(
	const std::size_t &batch_size,
	unsigned long &seed,
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const Widrow_Hoff &parameter,
	const std::size_t &stimulus_stride, const std::size_t &reservoir_stride, const std::size_t &prediction_stride,
	const std::size_t &stimulus_size, const std::size_t &reservoir_size, const std::size_t &prediction_size,
	const float &leak_rate, const float &initial_state_scale,
	float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	float **batched_w_ffwd, const std::size_t &batched_w_ffwd_rows, const std::size_t &batched_w_ffwd_cols, const std::size_t &batched_w_ffwd_strides,
	float **batched_u_ffwd, const std::size_t &batched_u_ffwd_rows, const std::size_t &batched_u_ffwd_cols, const std::size_t &batched_u_ffwd_strides,
	float **batched_x_in, const std::size_t &batched_x_in_rows, const std::size_t &batched_x_in_cols, const std::size_t &batched_x_in_strides,
	float **batched_w_in, const std::size_t &batched_w_in_rows, const std::size_t &batched_w_in_cols, const std::size_t &batched_w_in_strides,
	float **batched_u, const std::size_t &batched_u_rows, const std::size_t &batched_u_cols, const std::size_t &batched_u_strides,
	float **batched_p, const std::size_t &batched_p_rows, const std::size_t &batched_p_cols, const std::size_t &batched_p_strides,
	float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t &batched_w_ro_strides,
	float **batched_error, const std::size_t & batched_error_rows, const std::size_t &batched_error_cols, const std::size_t &batched_error_strides,
	const int *offsets, const int *durations, const std::size_t &repetitions,
	float *states_samples, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride
	);
template void update_model< false, true, Widrow_Hoff>(
	const std::size_t &batch_size,
	unsigned long &seed,
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const Widrow_Hoff &parameter,
	const std::size_t &stimulus_stride, const std::size_t &reservoir_stride, const std::size_t &prediction_stride,
	const std::size_t &stimulus_size, const std::size_t &reservoir_size, const std::size_t &prediction_size,
	const float &leak_rate, const float &initial_state_scale,
	float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	float **batched_w_ffwd, const std::size_t &batched_w_ffwd_rows, const std::size_t &batched_w_ffwd_cols, const std::size_t &batched_w_ffwd_strides,
	float **batched_u_ffwd, const std::size_t &batched_u_ffwd_rows, const std::size_t &batched_u_ffwd_cols, const std::size_t &batched_u_ffwd_strides,
	float **batched_x_in, const std::size_t &batched_x_in_rows, const std::size_t &batched_x_in_cols, const std::size_t &batched_x_in_strides,
	float **batched_w_in, const std::size_t &batched_w_in_rows, const std::size_t &batched_w_in_cols, const std::size_t &batched_w_in_strides,
	float **batched_u, const std::size_t &batched_u_rows, const std::size_t &batched_u_cols, const std::size_t &batched_u_strides,
	float **batched_p, const std::size_t &batched_p_rows, const std::size_t &batched_p_cols, const std::size_t &batched_p_strides,
	float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t &batched_w_ro_strides,
	float **batched_error, const std::size_t & batched_error_rows, const std::size_t &batched_error_cols, const std::size_t &batched_error_strides,
	const int *offsets, const int *durations, const std::size_t &repetitions,
	float *states_samples, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride
	);
template void update_model< false, false, Widrow_Hoff>(
	const std::size_t &batch_size,
	unsigned long &seed,
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const Widrow_Hoff &parameter,
	const std::size_t &stimulus_stride, const std::size_t &reservoir_stride, const std::size_t &prediction_stride,
	const std::size_t &stimulus_size, const std::size_t &reservoir_size, const std::size_t &prediction_size,
	const float &leak_rate, const float &initial_state_scale,
	float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	float **batched_w_ffwd, const std::size_t &batched_w_ffwd_rows, const std::size_t &batched_w_ffwd_cols, const std::size_t &batched_w_ffwd_strides,
	float **batched_u_ffwd, const std::size_t &batched_u_ffwd_rows, const std::size_t &batched_u_ffwd_cols, const std::size_t &batched_u_ffwd_strides,
	float **batched_x_in, const std::size_t &batched_x_in_rows, const std::size_t &batched_x_in_cols, const std::size_t &batched_x_in_strides,
	float **batched_w_in, const std::size_t &batched_w_in_rows, const std::size_t &batched_w_in_cols, const std::size_t &batched_w_in_strides,
	float **batched_u, const std::size_t &batched_u_rows, const std::size_t &batched_u_cols, const std::size_t &batched_u_strides,
	float **batched_p, const std::size_t &batched_p_rows, const std::size_t &batched_p_cols, const std::size_t &batched_p_strides,
	float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t &batched_w_ro_strides,
	float **batched_error, const std::size_t & batched_error_rows, const std::size_t &batched_error_cols, const std::size_t &batched_error_strides,
	const int *offsets, const int *durations, const std::size_t &repetitions,
	float *states_samples, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride
	);
template void update_model<true, true, Nothing>(
	const std::size_t &batch_size,
	unsigned long &seed,
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const Nothing &parameter,
	const std::size_t &stimulus_stride, const std::size_t &reservoir_stride, const std::size_t &prediction_stride,
	const std::size_t &stimulus_size, const std::size_t &reservoir_size, const std::size_t &prediction_size,
	const float &leak_rate, const float &initial_state_scale,
	float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	float **batched_w_ffwd, const std::size_t &batched_w_ffwd_rows, const std::size_t &batched_w_ffwd_cols, const std::size_t &batched_w_ffwd_strides,
	float **batched_u_ffwd, const std::size_t &batched_u_ffwd_rows, const std::size_t &batched_u_ffwd_cols, const std::size_t &batched_u_ffwd_strides,
	float **batched_x_in, const std::size_t &batched_x_in_rows, const std::size_t &batched_x_in_cols, const std::size_t &batched_x_in_strides,
	float **batched_w_in, const std::size_t &batched_w_in_rows, const std::size_t &batched_w_in_cols, const std::size_t &batched_w_in_strides,
	float **batched_u, const std::size_t &batched_u_rows, const std::size_t &batched_u_cols, const std::size_t &batched_u_strides,
	float **batched_p, const std::size_t &batched_p_rows, const std::size_t &batched_p_cols, const std::size_t &batched_p_strides,
	float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t &batched_w_ro_strides,
	float **batched_error, const std::size_t & batched_error_rows, const std::size_t &batched_error_cols, const std::size_t &batched_error_strides,
	const int *offsets, const int *durations, const std::size_t &repetitions,
	float *states_samples, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride
	);
template void update_model< true, false, Nothing >(
	const std::size_t &batch_size,
	unsigned long &seed,
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const Nothing &parameter,
	const std::size_t &stimulus_stride, const std::size_t &reservoir_stride, const std::size_t &prediction_stride,
	const std::size_t &stimulus_size, const std::size_t &reservoir_size, const std::size_t &prediction_size,
	const float &leak_rate, const float &initial_state_scale,
	float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	float **batched_w_ffwd, const std::size_t &batched_w_ffwd_rows, const std::size_t &batched_w_ffwd_cols, const std::size_t &batched_w_ffwd_strides,
	float **batched_u_ffwd, const std::size_t &batched_u_ffwd_rows, const std::size_t &batched_u_ffwd_cols, const std::size_t &batched_u_ffwd_strides,
	float **batched_x_in, const std::size_t &batched_x_in_rows, const std::size_t &batched_x_in_cols, const std::size_t &batched_x_in_strides,
	float **batched_w_in, const std::size_t &batched_w_in_rows, const std::size_t &batched_w_in_cols, const std::size_t &batched_w_in_strides,
	float **batched_u, const std::size_t &batched_u_rows, const std::size_t &batched_u_cols, const std::size_t &batched_u_strides,
	float **batched_p, const std::size_t &batched_p_rows, const std::size_t &batched_p_cols, const std::size_t &batched_p_strides,
	float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t &batched_w_ro_strides,
	float **batched_error, const std::size_t & batched_error_rows, const std::size_t &batched_error_cols, const std::size_t &batched_error_strides,
	const int *offsets, const int *durations, const std::size_t &repetitions,
	float *states_samples, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride
	);
template void update_model< false, true, Nothing>(
	const std::size_t &batch_size,
	unsigned long &seed,
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const Nothing &parameter,
	const std::size_t &stimulus_stride, const std::size_t &reservoir_stride, const std::size_t &prediction_stride,
	const std::size_t &stimulus_size, const std::size_t &reservoir_size, const std::size_t &prediction_size,
	const float &leak_rate, const float &initial_state_scale,
	float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	float **batched_w_ffwd, const std::size_t &batched_w_ffwd_rows, const std::size_t &batched_w_ffwd_cols, const std::size_t &batched_w_ffwd_strides,
	float **batched_u_ffwd, const std::size_t &batched_u_ffwd_rows, const std::size_t &batched_u_ffwd_cols, const std::size_t &batched_u_ffwd_strides,
	float **batched_x_in, const std::size_t &batched_x_in_rows, const std::size_t &batched_x_in_cols, const std::size_t &batched_x_in_strides,
	float **batched_w_in, const std::size_t &batched_w_in_rows, const std::size_t &batched_w_in_cols, const std::size_t &batched_w_in_strides,
	float **batched_u, const std::size_t &batched_u_rows, const std::size_t &batched_u_cols, const std::size_t &batched_u_strides,
	float **batched_p, const std::size_t &batched_p_rows, const std::size_t &batched_p_cols, const std::size_t &batched_p_strides,
	float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t &batched_w_ro_strides,
	float **batched_error, const std::size_t & batched_error_rows, const std::size_t &batched_error_cols, const std::size_t &batched_error_strides,
	const int *offsets, const int *durations, const std::size_t &repetitions,
	float *states_samples, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride
	);
template void update_model< false, false, Nothing>(
	const std::size_t &batch_size,
	unsigned long &seed,
	const hipStream_t &stream,
	const hipblasHandle_t &handle,
	const Nothing &parameter,
	const std::size_t &stimulus_stride, const std::size_t &reservoir_stride, const std::size_t &prediction_stride,
	const std::size_t &stimulus_size, const std::size_t &reservoir_size, const std::size_t &prediction_size,
	const float &leak_rate, const float &initial_state_scale,
	float **batched_incoming, const std::size_t &batched_incoming_rows, const std::size_t &batched_incoming_cols, const std::size_t &batched_incoming_strides,
	float **batched_expected, const std::size_t &batched_expected_rows, const std::size_t &batched_expected_cols, const std::size_t &batched_expected_strides,
	float **batched_w_ffwd, const std::size_t &batched_w_ffwd_rows, const std::size_t &batched_w_ffwd_cols, const std::size_t &batched_w_ffwd_strides,
	float **batched_u_ffwd, const std::size_t &batched_u_ffwd_rows, const std::size_t &batched_u_ffwd_cols, const std::size_t &batched_u_ffwd_strides,
	float **batched_x_in, const std::size_t &batched_x_in_rows, const std::size_t &batched_x_in_cols, const std::size_t &batched_x_in_strides,
	float **batched_w_in, const std::size_t &batched_w_in_rows, const std::size_t &batched_w_in_cols, const std::size_t &batched_w_in_strides,
	float **batched_u, const std::size_t &batched_u_rows, const std::size_t &batched_u_cols, const std::size_t &batched_u_strides,
	float **batched_p, const std::size_t &batched_p_rows, const std::size_t &batched_p_cols, const std::size_t &batched_p_strides,
	float **batched_x_res, const std::size_t &batched_x_res_rows, const std::size_t &batched_x_res_cols, const std::size_t &batched_x_res_strides,
	float **batched_x_ro, const std::size_t &batched_x_ro_rows, const std::size_t &batched_x_ro_cols, const std::size_t &batched_x_ro_strides,
	float **batched_w_ro, const std::size_t &batched_w_ro_rows, const std::size_t &batched_w_ro_cols, const std::size_t &batched_w_ro_strides,
	float **batched_error, const std::size_t & batched_error_rows, const std::size_t &batched_error_cols, const std::size_t &batched_error_strides,
	const int *offsets, const int *durations, const std::size_t &repetitions,
	float *states_samples, const std::size_t &states_rows, const std::size_t &states_cols, const std::size_t &states_stride
	);

